//first cuda program
//Hitender Prakash


#include <hip/hip_runtime.h>
#include <stdio.h>

//define gpu kernel

__global__ void square(double *d_out, double *d_in){
	int pos=threadIdx.x;
	d_out[pos]=d_in[pos]*d_in[pos];
}

int main(int argc, char **argv){
	if(argc <2 ||argc >2){
		printf("\nUsage: sqaure <size of array>");
		exit(0);
	}

	int siz=atoi(argv[1]);
	
	double *d_in, *d_out, *h_in, *h_out;
	
	h_in=(double *)malloc(siz*sizeof(double));
	h_out=(double *)malloc(siz*sizeof(double));
	
	for(int i=0;i<siz;i++){
		h_in[i]=i+1.0;
		h_out[i]=0.0;
	}
	
	//allocate space on GPU
	hipMalloc((void**)&d_in, (size_t)siz*sizeof(double));
	int err= hipGetLastError();
	
	hipMalloc((void**)&d_out, (size_t)siz*sizeof(double));
	
	//copy from host to device
	hipMemcpy(d_in, h_in, siz*sizeof(double), hipMemcpyHostToDevice);
	square<<<1,siz>>>(d_out,d_in);
	hipMemcpy(h_out, d_out, siz*sizeof(double), hipMemcpyDeviceToHost);
	
	printf("\nBelow is the processed square values: ");
	for(int i=0;i<siz;i++){
		printf("\n%lf ----> %lf",h_in[i],h_out[i]);
	}	
	printf("\nLast cuda error in malloc: %d",err);
	printf("\n");
	return 0;
}
